#include "hip/hip_runtime.h"
#include <iostream>
#include "TheEmployeesSalary.h"
#include "hip/hip_runtime.h"
#include ""

// Note that primitive types (like double) can be passed by value in CUDA
__global__ void gpu_salary_incrementer(const double* original_salary, double* new_salary, int size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size) {
        new_salary[i] = original_salary[i] * 1.15 + 5000;
    }
}

hipError_t device_function_calls(double* cpu_TheArrayOfNewSalaries, double* gpu_TheArrayOfNewSalaries, double* d_original_salary, double* d_new_salary, int size) {
    int threads_per_block = 256;
    int blocks_per_grid = (size + threads_per_block - 1) / threads_per_block;

    hipError_t cuda_status = hipSetDevice(0);
    if (cuda_status != hipSuccess) {
        std::cerr << "hipSetDevice failed! Do you have a CUDA-capable GPU installed?" << hipGetErrorString(cuda_status) << std::endl;
        return cuda_status;
    }

    // 1. Allocation device memory
    cuda_status = hipMalloc((void**)&d_original_salary, size * sizeof(double));
    if (cuda_status != hipSuccess) {
        std::cerr << "hipMalloc failed for d_original_salary!" << hipGetErrorString(cuda_status) << std::endl;
        return cuda_status;
    }

    cuda_status = hipMalloc((void**)&d_new_salary, size * sizeof(double));
    if (cuda_status != hipSuccess) {
        std::cerr << "hipMalloc failed for d_new_salary!" << hipGetErrorString(cuda_status) << std::endl;
        hipFree(d_original_salary);
        return cuda_status;
    }

    // 2. Copy data from host to device
    cuda_status = hipMemcpy(d_original_salary, TheArrayOfSalaries, size * sizeof(double), hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess) {
        std::cerr << "hipMemcpy failed for d_original_salary!" << hipGetErrorString(cuda_status) << std::endl;
        goto ErrorHandler;
    }

    // 3. Kernel launch
    gpu_salary_incrementer<<<blocks_per_grid, threads_per_block>>>(d_original_salary, d_new_salary, size);
    cuda_status = hipDeviceSynchronize();
    if (cuda_status != hipSuccess) {
        std::cerr << "hipDeviceSynchronize failed!" << hipGetErrorString(cuda_status) << std::endl;
        goto ErrorHandler;
    }

    // 4. Copy data from device to host
    cuda_status = hipMemcpy(gpu_TheArrayOfNewSalaries, d_new_salary, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cuda_status != hipSuccess) {
        std::cerr << "cudaMemCpy failed for Device to Host!" << hipGetErrorString(cuda_status) << std::endl;
        goto ErrorHandler;
    }

    // 5. Free device memory)
    cuda_status = hipFree(d_original_salary);
    if (cuda_status != hipSuccess) {
        std::cerr << "hipFree failed for d_original_salary!" << hipGetErrorString(cuda_status) << std::endl;
        hipFree(d_new_salary);
    }

    cuda_status = hipFree(d_new_salary);
    if (cuda_status != hipSuccess) {
        std::cerr << "hipFree failed for d_new_salary!" << hipGetErrorString(cuda_status) << std::endl;
    }

    cuda_status = hipGetLastError(); // Check for any kernel launch errors
    if (cuda_status != hipSuccess) {
        std::cerr << "Error Detected!" << hipGetErrorString(cuda_status) << std::endl;
    }
    return cuda_status;

    ErrorHandler:
        hipFree(d_original_salary);
        hipFree(d_new_salary);
        return cuda_status;
}

int main() {
    int size = sizeof(TheArrayOfSalaries) / sizeof(double);

    // CPU Computation for Reference
    double cpu_TheArrayOfNewSalaries[size] = {0}; // Define an array to hold new salaries, all 0's
    cpu_salary_incrementer(TheArrayOfSalaries, cpu_TheArrayOfNewSalaries, size);

    // GPU Computation
    double gpu_TheArrayOfNewSalaries[size] = {0};
    double* d_original_salary;
    double* d_new_salary;
    hipError_t cuda_status = device_function_calls(cpu_TheArrayOfNewSalaries, gpu_TheArrayOfNewSalaries, d_original_salary, d_new_salary, size);
    if (cuda_status != hipSuccess) {
        std::cerr << "Device Function Calls Failed! CUDA error: " << hipGetErrorString(cuda_status) << std::endl;
        return 1;
    }

    // Compare
    bool comparison_result = compare_results(cpu_TheArrayOfNewSalaries, gpu_TheArrayOfNewSalaries, size);
    std::cout << "Comparison result: " << (comparison_result ? "Match" : "Mismatch") << std::endl;
    
    return 0;
}