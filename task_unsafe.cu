#include "hip/hip_runtime.h"
#include <iostream>
#include "TheEmployeesSalary.h"
#include "hip/hip_runtime.h"
#include ""


// Note that primitive types (like double) can be passed by value in CUDA
__global__ void gpu_salary_incrementer(const double* original_salary, double* new_salary, int size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size) {
        new_salary[i] = original_salary[i] * 1.15 + 5000;
    }
}

int main() {
    int size = sizeof(TheArrayOfSalaries) / sizeof(double);

    // CPU Computation for Reference
    double cpu_TheArrayOfNewSalaries[size] = {0};
    cpu_salary_incrementer(TheArrayOfSalaries, cpu_TheArrayOfNewSalaries, size);

    // GPU Computation
    // 1. Allocation device memory
    double* d_original_salary;
    double* d_new_salary;
    hipMalloc((void**)&d_original_salary, size * sizeof(double));
    hipMalloc((void**)&d_new_salary, size * sizeof(double));

    // 2. Copy data from host to device
    hipMemcpy(d_original_salary, TheArrayOfSalaries, size * sizeof(double), hipMemcpyHostToDevice);

    // 3. Kernel launch
    int threads_per_block = 256;
    int blocks_per_grid = (size + threads_per_block - 1) / threads_per_block;
    gpu_salary_incrementer<<<blocks_per_grid, threads_per_block>>>(d_original_salary, d_new_salary, size);
    hipDeviceSynchronize();

    // 4. Copy data from device to host
    double gpu_TheArrayOfNewSalaries[size] = {0};
    hipMemcpy(gpu_TheArrayOfNewSalaries, d_new_salary, size * sizeof(double), hipMemcpyDeviceToHost);

    // 5. Free device memory)
    hipFree(d_original_salary);
    hipFree(d_new_salary);

    // Compare
    bool comparison_result = compare_results(cpu_TheArrayOfNewSalaries, gpu_TheArrayOfNewSalaries, size);
    std::cout << "Comparison result: " << (comparison_result ? "Match" : "Mismatch") << std::endl;
    
    return 0;
}